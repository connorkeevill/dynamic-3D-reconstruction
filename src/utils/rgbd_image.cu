#include "hip/hip_runtime.h"
// Copyright 2018 Emanuele Palazzolo (emanuele.palazzolo@uni-bonn.de), Cyrill Stachniss, University of Bonn
#include "utils/rgbd_image.h"

namespace refusion {

	RgbdImage::~RgbdImage()
	{
		hipDeviceSynchronize();
		hipFree(rgb_);
		hipFree(depth_);
	}

	void RgbdImage::Init(const RgbdSensor &sensor)
	{
		sensor_ = sensor;
		hipMallocManaged(&rgb_, sizeof(uchar3) * sensor_.rows * sensor.cols);
		hipMallocManaged(&depth_, sizeof(float) * sensor_.rows * sensor.cols);
		hipDeviceSynchronize();
	}

	__host__ __device__ inline float3 RgbdImage::GetPoint3d(int u, int v) const
	{
		float3 point;
		point.z = depth_[v * sensor_.cols + u];
		point.x = (static_cast<float>(u) - sensor_.cx) * point.z / sensor_.fx;
		point.y = (static_cast<float>(v) - sensor_.cy) * point.z / sensor_.fy;
		return point;
	}

	__host__ __device__ inline float3 RgbdImage::GetPoint3d(int i) const
	{
		int v = i / sensor_.cols;
		int u = i - sensor_.rows * v;
		return GetPoint3d(u, v);
	}

}  // namespace refusion
